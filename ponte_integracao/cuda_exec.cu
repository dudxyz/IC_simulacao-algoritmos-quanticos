// cuda_exec.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define BLOCK_SIZE 256

__global__ void gpu_matrix_vector_mult(hipFloatComplex *matrix, hipFloatComplex *vector, hipFloatComplex *result, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
        for (int j = 0; j < cols; j++) {
            sum = hipCaddf(sum, hipCmulf(matrix[row * cols + j], vector[j]));
        }
        result[row] = sum;
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Uso: ./cuda_exec <dimensao>\n");
        return 1;
    }

    int dim = atoi(argv[1]);
    int rows = dim, cols = dim;
    size_t size = rows * cols * sizeof(hipFloatComplex);
    size_t vecSize = cols * sizeof(hipFloatComplex);

    hipFloatComplex *h_matrix = (hipFloatComplex*)malloc(size);
    hipFloatComplex *h_vector = (hipFloatComplex*)malloc(vecSize);
    hipFloatComplex *h_result = (hipFloatComplex*)malloc(vecSize);

    FILE *f = fopen("matrix.bin", "rb");
    fread(h_matrix, sizeof(hipFloatComplex), rows * cols, f);
    fclose(f);

    f = fopen("vector.bin", "rb");
    fread(h_vector, sizeof(hipFloatComplex), cols, f);
    fclose(f);

    hipFloatComplex *d_matrix, *d_vector, *d_result;
    hipMalloc((void**)&d_matrix, size);
    hipMalloc((void**)&d_vector, vecSize);
    hipMalloc((void**)&d_result, vecSize);

    hipMemcpy(d_matrix, h_matrix, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector, h_vector, vecSize, hipMemcpyHostToDevice);

    int gridSize = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    gpu_matrix_vector_mult<<<gridSize, BLOCK_SIZE>>>(d_matrix, d_vector, d_result, rows, cols);

    hipMemcpy(h_result, d_result, vecSize, hipMemcpyDeviceToHost);

    FILE *out = fopen("result.bin", "wb");
    fwrite(h_result, sizeof(hipFloatComplex), cols, out);
    fclose(out);

    free(h_matrix); free(h_vector); free(h_result);
    hipFree(d_matrix); hipFree(d_vector); hipFree(d_result);

    return 0;
}
